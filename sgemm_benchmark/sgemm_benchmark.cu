#include "hip/hip_runtime.h"
#include <iostream>
#include <benchmark/benchmark.h>
#include <hipblas.h>
#include <hiprand.h>
#include <sys/types.h>

using namespace std;

static void cuda_mul_matrix (benchmark::State& s)
{
	int N = 1 << s.range(0);

	size_t bytes = N * N * sizeof(float);

    // Allocate memory on the host side
    float *host_a = new float[N * N];
    float *host_b = new float[N * N];
    float *host_c = new float[N * N];

    // Allocate memory on the device side
    float *dev_a;
    float *dev_b;
    float *dev_c;

    hipMalloc(&dev_a, bytes);
    hipMalloc(&dev_b, bytes);
    hipMalloc(&dev_c, bytes);

	// generate random numbers using a system clock seed
	hiprandGenerator_t prng; 
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	// use the random number generator to generate values for our device pointers
	hiprandGenerateUniform(prng, dev_a, N * N);
	hiprandGenerateUniform(prng, dev_b, N * N);
	hiprandGenerateUniform(prng, dev_c, N * N);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	const float alpha = 2.0f;
	const float beta = 3.0f;
	while(s.KeepRunning())
	{
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, dev_a, N, dev_b, N, &beta, dev_c, N);
	}

	hipblasGetVector(N * N, sizeof(float), dev_c, 1, host_c, 1);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	delete[] host_a;
	delete[] host_b;
	delete[] host_c;
}

BENCHMARK(cuda_mul_matrix) -> DenseRange(6, 10) -> Unit(benchmark::kMicrosecond);

static void naive_mul_matrix (benchmark::State& s)
{
	int N = 1 << s.range(0);

    // Allocate memory on the host side
    float *host_a = new float[N * N];
    float *host_b = new float[N * N];
    float *host_c = new float[N * N];

    for (int i = 0; i < N * N; i++)
    {
        host_a[i] = rand() % 100;
        host_b[i] = rand() % 100;
        host_c[i] = rand() % 100;
	}
	
	const float alpha = 2.0f;
	const float beta = 3.0f;

	while(s.KeepRunning())
	{
		for (int i = 0; i < N; i++)
		{
			for (int j = 0; j < N; j++)
			{
				int temp;
				for (int k = 0; k < N; k++)
				{
					temp += (alpha * host_a[i*N + k]) * host_b[k*N + j];
				}
				host_c[i*N + j] = temp + (beta * host_c[i*N +j]);
			}
		}
		benchmark::DoNotOptimize(host_c);
	}

	delete[] host_a;
	delete[] host_b;
	delete[] host_c;
}

BENCHMARK(naive_mul_matrix) -> DenseRange(6, 10) -> Unit(benchmark::kMicrosecond);

BENCHMARK_MAIN();